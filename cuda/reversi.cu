#include "reversi.cuh"
#include "../gameplay/reversi.h"

#include <err.h>
#include <stdio.h>

/**
 * A Warp is a logical unit that shares a pc
 * Keep thread blocks as large as possible without increasing divergence
 */

 __host__ board create_board_cuda(uint8_t starting_player, uint8_t height, uint8_t width) {
    board b;
    hipMallocManaged(&b, sizeof(board_str));
    if(!b) err(1, "Memory Error while allocating the board\n");
    b->player = starting_player;
    b->height = height;
    b->width = width;

    hipMallocManaged(&b->board, sizeof(uint8_t) * ((height * width) >> 2));
    memset(b->board, 0, sizeof(uint8_t) * ((height * width) >> 2));
    if(!b->board) err(1, "Memory Error while allocating board's board array\n");

    /* <1 byte> 
     * +-+-+-+-+-+-+-+-+
     * | | | | | | | | | <-- 2 bytes | Byte 0,1
     * +-+-+-+-+-+-+-+-+
     * | | | | | | | | | <-- Byte 2,3
     * +-+-+-+-+-+-+-+-+
     * | | | | | | | | | <-- Byte 4,5
     * +-+-+-+-+-+-+-+-+
     * | | | |2|1| | | | <-- Byte 6,7
     * +-+-+-+-+-+-+-+-+
     * | | | |1|2| | | | <-- Byte 8,9
     * +-+-+-+-+-+-+-+-+
     * | | | | | | | | | <-- Byte 10,11
     * +-+-+-+-+-+-+-+-+
     * | | | | | | | | | <-- Byte 12,13
     * +-+-+-+-+-+-+-+-+
     * | | | | | | | | | <-- Byte 14,15
     * +-+-+-+-+-+-+-+-+
     */

    /** To create the starting position we need to fill the bits as such:
     * +--+--+--+--+--+--+--+--+
     * |00|00|00|10|01|00|00|00| <-- 2,64 for byte 6 and 7
     * +--+--+--+--+--+--+--+--+
     * |00|00|00|01|10|00|00|00| <-- 1,128 for byte 8 and 9
     * +--+--+--+--+--+--+--+--+
     */

    board_put(b, (height >> 1) - 1, (width >> 1) - 1, 2);
    board_put(b, (height >> 1) - 1, width >> 1, 1);
    board_put(b, height >> 1, (width >> 1) - 1, 1);
    board_put(b, height >> 1, width >> 1, 2);

    return b;
 }

 __host__ void destroy_board_cuda(board b) {
     hipFree(b->board);
     hipFree(b);
 }

 __host__ __device__ uint8_t board_get_cuda(board b, uint8_t row, uint8_t column) {
    if(b) {
        // printf("Fetching board square\n");
        uint8_t total_bit = (row * (b->width << 1)) + (column << 1), 
                byte = total_bit >> 3, 
                bit = total_bit % 8;

        return ((192 >> bit) & b->board[byte]) >> (6 - bit);
    }
    return 3;
}

__host__ __device__ void board_put_cuda(board b, uint8_t row, uint8_t column, uint8_t player) {
    if(b) {
        uint8_t total_bit = (row * (b->width << 1)) + (column << 1), 
                byte = total_bit >> 3, 
                bit = total_bit % 8, 
                bph = 192 >> bit;

        b->board[byte] = (b->board[byte] | bph) ^ bph;

        if(player) b->board[byte] |= ((player == 1) ? 64 : 128) >> bit;
    }
}

__host__ __device__ uint8_t board_is_legal_move_cuda(board b, uint8_t row, uint8_t column) {
    if(b && row < b->height && column < b->width) {
        if(!board_get_cuda(b, row, column)) {

            // printf("Managed to check a board\n");

            // Check each of the 8 directions going out from the requested coordinate
            // Keep track of how many captures we have
            int8_t counts = 0, cr, cc, count, bv, operating;
            for(int8_t rd = -1; rd < 2; rd++) {
                for(int8_t cd = -1; cd < 2; cd++) {
                    // Avoid infinite loop when rd=cd=0
                    if(!rd && !cd) continue;

                    // Take a step in the current direction
                    cr = row + rd;
                    cc = column + cd;

                    count = 0;
                    while(cr < b->height && cc < b->width) {
                        bv = board_get_cuda(b, cr, cc);
                        if(bv && bv != b->player) {
                            // There is a possible capture
                            count++;

                            // Take another step in the current direction
                            cr += rd;
                            cc += cd;

                            if((cr == b->height && rd) ||
                               (cr < 0 && rd == -1) ||
                               (cc == b->width && cd) ||
                               (cc < 0 && cd < 0)) {
                                   // We hit the edge of the board, this is not a capture
                                   count = 0;
                                   break;
                               }
                        }
                        else {
                            if(!bv)
                                // If we had any captures, they are in vain because our color isn't at the other end.
                                count = 0;
                            break;
                        }
                    }
                    counts += count;
                }
            }

            // Return true if we capture at least 1 piece
            return counts > 0;
        }
    }

    // Either the board pointer was empty, or the space was already filled.
    return 0;
}

__host__ __device__ void clone_into_board_cuda(board src, board dest) {
    if(src && dest) {
        dest->height = src->height;
        dest->width = src->width;
        dest->player = src->player;
        for(uint8_t i = 0; i < ((src->height * src->width) >> 2); i++) dest->board[i] = src->board[i];
    }
}

__host__ __device__ void board_place_piece_cuda(board b, uint8_t row, uint8_t column) {
    if(b && row >= 0 && row < b->height && column >= 0 && column < b->width) {
        board_put_cuda(b, row, column, b->player);
        int8_t cr, cc, bv;
        uint8_t count;
        for(int8_t rd = -1; rd < 2; rd++) {
            for(int8_t cd = -1; cd < 2; cd++) {
                // Avoid infinite loop when rd=cd=0
                if(!rd && !cd) continue;
    
                // Take a step in the current direction
                cr = row + rd;
                cc = column + cd;
    
                count = 0;
                while(cr >= 0 && cr < b->height && cc >= 0 && cc < b->width) {
                    bv = board_get_cuda(b, cr, cc);
                    if(bv && bv != b->player) {
                        // There is a possible capture
                        count++;
    
                        // Take another step in the current direction
                        cr += rd;
                        cc += cd;
    
                        if((cr == b->height && rd) ||
                            (cr < 0 && rd == -1) ||
                            (cc == b->width && cd) ||
                            (cc < 0 && cd < 0)) {
                                // We hit the edge of the board, this is not a capture
                                count = 0;
                                break;
                            }
                    }
                    else {
                        if(!bv)
                            // If we had any captures, they are in vain because our color isn't at the other end.
                            count = 0;
                        break;
                    }
                }
                
                if(count > 0) {
                    cr = row + rd;
                    cc = column + cd;
                    bv = board_get_cuda(b, cr, cc);
    
                    while(bv && bv != b->player) {
                        board_put_cuda(b, cr, cc, b->player);
                        cr += rd;
                        cc += cd;
                        bv = board_get_cuda(b, cr, cc);
                    }
                }
            }
        }
    
        // Flip the player to the opponent
        b->player = (b->player == 1) ? 2 : 1;
    }
}